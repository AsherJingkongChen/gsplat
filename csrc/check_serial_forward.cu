#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <iostream>
#include <iomanip>
#include <math.h>
#include <vector>

#include "forward.cuh"
#include "helpers.cuh"

#include "debug_utils.h"

#include "reference/cuda_rasterizer/forward.cu"

#define DOCTEST_CONFIG_IMPLEMENT
#include <doctest/doctest.h>

float random_float() { return (float)std::rand() / RAND_MAX; }

float4 random_quat() {
    float u = random_float();
    float v = random_float();
    float w = random_float();
    return {
        sqrt(1.f - u) * sin(2.f * (float)M_PI * v),
        sqrt(1.f - u) * cos(2.f * (float)M_PI * v),
        sqrt(u) * sin(2.f * (float)M_PI * w),
        sqrt(u) * cos(2.f * (float)M_PI * w)};
}

void compare_cov2d_forward(){
    // TODO: test with more than one point and varying cov3d/viewmat
    int num_points = 1;
    float fx = 1;
    float fy = 1;
    const int W = 256;
    const int H = 256;
    float tan_fovx = 0.5 * W / fx;
    float tan_fovy = 0.5 * H / fy;
    float viewmat[] = {
        1.f, 0.f, 0.f, 0.f,
        0.f, 1.f, 0.f, 0.f,
        0.f, 0.f, 1.f, 8.f,
        0.f, 0.f, 0.f, 1.f};
    float viewmatColumnMajor[] = {
        1.0f, 0.0f, 0.0f, 0.0f,
        0.0f, 1.0f, 0.0f, 0.0f,
        0.0f, 0.0f, 1.0f, 0.0f,
        0.0f, 0.0f, 8.0f, 1.0f
    };
    float cov3d[]{
        1.f, 0.f, 0.f, 
        1.f, 0.f, 1.f
    };
    float3 mean = {1.f, 1.f, 1.f};

    // diff rast
    float3 diff_rast_cov2d = project_cov3d_ewa(mean,cov3d, viewmat, fx,fy);
    std::cout<<"diff rast cov2d: "<<diff_rast_cov2d.x<<" "<<diff_rast_cov2d.y<<" "<<diff_rast_cov2d.z<<std::endl; 
    
    // ref rast
    float3 res = computeCov2D(mean, fx, fy, tan_fovx, tan_fovy, cov3d, viewmatColumnMajor);
    std::cout<<"ref rast cov2d:  "<<res.x<<" "<<res.y<<" "<<res.z<<std::endl;


    const std::vector<std::pair<float,float>> cov2d_data {
        {diff_rast_cov2d.x, res.x},
        {diff_rast_cov2d.y, res.y},
        {diff_rast_cov2d.z, res.z},
    };
    print_errors(cov2d_data, "diff_rast_cov2d (cov2d_forward)");
}

void compare_scale_rot_to_cov3d(){
    // TODO: make it work with more than one point
    int num_points = 1;
    float3 scale = {random_float(), random_float(), random_float()};
    float glob_scale = random_float();

    float4 quat = random_quat();

    float* cov3d = new float[6 * num_points];

    // diff rast
    scale_rot_to_cov3d(scale, glob_scale, quat, cov3d);   
    std::cout << "diff rast cov3d: " << cov3d[0]<<" "<<cov3d[1]<<" "<<cov3d[2]<<std::endl;

    // ref rast
    float* ref_cov3d = new float[6 * num_points];
    computeCov3D(glm::vec3({scale.x,scale.y,scale.z}), glob_scale, glm::vec4({quat.x, quat.y, quat.z, quat.w}), ref_cov3d);
    std::cout << "ref rast cov3d: " <<ref_cov3d[0]<<" "<<ref_cov3d[1]<<" "<<ref_cov3d[2]<<std::endl;


    std::vector<std::pair<float,float>> cov3d_data;
    for (int i = 0; i < 3; ++i)   // Jonathan: is this supposed to just go to 3?
        cov3d_data.push_back({cov3d[i], ref_cov3d[i]});

    print_errors(cov3d_data, "diff rast cov3d (scale_rot_to_cov3d)");
}

// doctest example code start
int factorial(int number) { return number <= 1 ? number : factorial(number - 1) * number; }

TEST_CASE("testing the factorial function") {
    CHECK(factorial(1) == 1);
    CHECK(factorial(2) == 2);
    CHECK(factorial(3) == 6);
    CHECK(factorial(10) == 3628800);
    CHECK(factorial(0) == -123);    // intentional failure; remove me to see
                                    // what passing tests looks like :)
}
// doctest example code end

int main(){

    compare_cov2d_forward();
    compare_scale_rot_to_cov3d();

    // todo: write doctest tests
    doctest::Context context;

    int results = context.run();    // runs tests

    return results;
}
