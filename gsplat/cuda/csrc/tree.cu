#include "hip/hip_runtime.h"
#include "bindings.h"
#include "helpers.cuh"
#include "third_party/glm/glm/glm.hpp"
#include "third_party/glm/glm/gtc/type_ptr.hpp"
#include "utils.cuh"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace cg = cooperative_groups;

// /*
// Given all the leaf nodes of the tree, as well as the tree structure, we want to
// find a cut in the tree to locate the nodes (internal or leaf) that satisfy a
// specific condition (e.g., node above the cut is larger than the cut and node
// below the cut is smaller than the cut).

// The tree is currently set as a 2^N branching factor tree. (e.g., N=3 means there
// are 8 children for each node)

// Tutorial:
// https://developer.nvidia.com/blog/using-cuda-warp-level-primitives/
// */
// template <uint32_t DIM>
// __global__ void
// tree_cut_kernel(const uint32_t N,      // 2^N is the branching factor of the tree
//                 const uint32_t n_leaf, // number of leaf nodes in the tree
//                 const float *__restrict__ leaf_data, // [n_leaf, DIM]
//                 const bool *__restrict__ leaf_mask,  // [n_leaf]
//                 const float cut,                     // the cut value
//                 // outputs
//                 float *__restrict__ selected_data, // [n_leaf]
//                 bool *__restrict__ selected_mask   // [n_leaf]
// ) {
//     // The number of threads is the same as number of leaves.

//     // In the first step, each thread will check if the leaf node satisfies the
//     // condition (larger than the cut). If it does, it will set the mask to
//     // true, and write out the data (but the thread still stays alive).

//     // In the second step, all children (every 2^N threads) of the same parent
//     // will reduce the data into the 0-th lane of the every 2^N threads, to
//     // calculate the data of the parent node. (using __shfl_down_sync)

//     // In the third step, we check if the parent node satisfies the condition.
//     // This only needs to be done by the 0-th lane of every 2^N threads, as it
//     // is where the parent node data is calculated. If the parent node is larger
//     // than the cut, we send the signal to all the children (2^N threads). The
//     // children which are smaller than the cut will be selected and write out
//     // the data and mask. If the parent node is smaller than the cut, we know
//     // all the children are smaller than the cut, so we don't need to send the
//     // signal to the children. (using __shfl_sync)

//     // In the fourth step, we repeat the second step to calculate the
//     // grandparent node data (every 2^(N+1) threads), and repeat the following
//     // steps.

//     // The process will be repeated until we reach the root node. (all threads
//     // are reduced to 0-th lane)

//     uint32_t idx = cg::this_grid().thread_rank();
//     if (idx >= n_leaf)
//         return;

//     // Step 1

//     // Read the data from global memory
//     float data[DIM] = {0.f};
//     if (leaf_mask[idx]) {
// #pragma unroll
//         for (uint32_t i = 0; i < DIM; i++) {
//             data[i] = leaf_data[idx * DIM + i];
//         }
//     }

//     // Check if the leaf node satisfies the condition.
//     // In this example code we use the first dimension of the data to compare
//     // with the cut. In the hierachical GS case we need to project the GS to
//     // image plane and compare the projected radius with the cut.
//     bool is_larger = data[0] > cut;
//     printf("thread %d: is_larger %d\n", idx, is_larger);

//     // Write out the data and mask if the leaf node satisfies the condition.
//     if (is_larger) {
//         printf("thread %d: selected\n", idx);
// #pragma unroll
//         for (uint32_t i = 0; i < DIM; i++) {
//             selected_data[idx * DIM + i] = data[i];
//         }
//         selected_mask[idx] = true;
//     }

//     // Step 2

//     // Copy the data to parent_data. This is necessary as the following
//     // __shfl_down_sync operation will overwrite the data it applies to.
//     float parent_data[DIM] = {0.f};
// #pragma unroll
//     for (int32_t i = 0; i < DIM; i++) {
//         parent_data[i] = data[i];
//     }

//     // Calculate the parent node data via __shfl_down_sync (every 2^N threads)
// #pragma unroll
//     for (int32_t i = 0; i < DIM; i++) {
//         // e.g., N == 3: branching factor is 8, then we need to
//         // __shfl_down_sync(m, v, 4)
//         // __shfl_down_sync(m, v, 2)
//         // __shfl_down_sync(m, v, 1)
//         // to get the parent node data (simply a sum op in this example).
//         for (int32_t j = N - 1; j >= 0; j--) {
//             float v = __shfl_down_sync(0xFFFFFFFF, parent_data[i], 1 << j);
//             printf("thread %d: j %d, v %f, parent_data[%d] %f\n", idx, j, v, i,
//                    parent_data[i]);
//             parent_data[i] += v;
//         }
//     }
//     printf("thread %d: parent_data[0] %f\n", idx, parent_data[0]);

//     // Step 3

//     // Check if the parent node satisfies the condition.
//     bool is_parent_larger = false;
//     if (idx % (1 << N) == 0) {
//         is_parent_larger = parent_data[0] > cut;
//         printf("thread %d (parent): is_parent_larger: %d\n", idx, is_parent_larger);
//     }

//     // Send the is_parent_larger signal from the 0-th lane of every 2^N threads
//     // to all the 2^N threads, using __shfl_sync.
//     uint32_t lane_id_parent = idx / (1 << N) * (1 << N);
//     is_parent_larger = __shfl_sync(0xFFFFFFFF, is_parent_larger, lane_id_parent);
//     printf("thread %d (children): is_parent_larger: %d\n", idx, is_parent_larger);

//     // Step 4

//     // See if the children satisfy the condition.
//     if (is_parent_larger & !is_larger) {
//         // Write out the data and mask if the child node satisfies the
//         // condition.
//         printf("thread %d: selected\n", idx);
// #pragma unroll
//         for (uint32_t i = 0; i < DIM; i++) {
//             selected_data[idx * DIM + i] = data[i];
//         }
//         selected_mask[idx] = true;
//     }

//     //     else {
//     //         // If not satisfied, then update the data as the parent data, as
//     //         this lane
//     //         // now represents the parent node.
//     //         printf("thread %d: not selected. set parent data as data\n",
//     //         idx);
//     // #pragma unroll
//     //         for (uint32_t i = 0; i < DIM; i++) {
//     //             data[i] = parent_data[i];
//     //         }
//     //     }

//     // Repeat the process until we reach the root node.
//     // To be implemented.
// }

// std::tuple<torch::Tensor, torch::Tensor> tree_cut_tensor(
//     const torch::Tensor &leaf_data,
//     const torch::Tensor &leaf_mask,
//     const int32_t branch_factor,
//     const float cut
// ) {
//     // Check the input tensor
//     TORCH_CHECK(leaf_data.dim() == 2, "leaf_data must be 2D tensor");
//     TORCH_CHECK(leaf_mask.dim() == 1, "leaf_mask must be 1D tensor");
//     TORCH_CHECK(
//         leaf_data.size(0) == leaf_mask.size(0),
//         "leaf_data and leaf_mask must have the same size"
//     );

//     // Get the number of leaf nodes and the dimension of the data
//     const uint32_t n_leaf = leaf_data.size(0);
//     const uint32_t DIM = leaf_data.size(1);
//     const uint32_t N = static_cast<uint32_t>(std::log2(branch_factor));

//     // Allocate the output tensor
//     torch::Tensor selected_data =
//         torch::zeros({n_leaf, DIM}, leaf_data.options());
//     torch::Tensor selected_mask = torch::zeros({n_leaf},
//     leaf_mask.options());

//     if (n_leaf) {
//         at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
//         switch (DIM) {
//         case 1:
//             tree_cut_kernel<1>
//                 <<<(n_leaf + N_THREADS - 1) / N_THREADS,
//                    N_THREADS,
//                    0,
//                    stream>>>(
//                     N,
//                     n_leaf,
//                     leaf_data.data_ptr<float>(),
//                     leaf_mask.data_ptr<bool>(),
//                     cut,
//                     selected_data.data_ptr<float>(),
//                     selected_mask.data_ptr<bool>()
//                 );
//             break;
//         default:
//             TORCH_CHECK(false, "Unsupported dimension of the data");
//         }
//     }

//     return std::make_tuple(selected_data, selected_mask);
// }

// BR = 2^N which is the branching factor of the tree
template <uint32_t DIM, uint32_t BR>
__global__ void
tree_cut2_kernel(const uint32_t n_leaf, // number of leaf nodes in the tree
                 const float *__restrict__ leaf_data, // [n_leaf, DIM]
                 const bool *__restrict__ leaf_mask,  // [n_leaf]
                 const float cut,                     // the cut value
                 // outputs
                 float *__restrict__ selected_data, // [n_leaf]
                 bool *__restrict__ selected_mask   // [n_leaf]
) {
    // The number of threads is n_leaf / 2^N.

    // Each block contains N_THREADS (e.g., 256) threads.
    // Each thread will process 2^N leaf nodes.
    // So the shared memory (all threads within a block have access to) will
    // store N_THREADS * 2^N nodes.

    extern __shared__ int sm[];
    // [N_THREADS * BR * DIM]
    float *sm_data = reinterpret_cast<float *>(sm);
    // [N_THREADS * BR]
    bool *sm_mask = reinterpret_cast<bool *>(&sm_data[N_THREADS * BR * DIM]);

    float block_total_data[DIM] = {0.f};
    for (uint32_t block_col = 0; block_col < n_leaf; block_col += BR * N_THREADS) {

        // Load data into shared memory (BR nodes per thread).
        for (uint32_t i = 0; i < BR; i++) {
            uint32_t offset = threadIdx.x + i * N_THREADS;
            uint32_t col = block_col + offset;

            if (col < n_leaf) {
                // read from global memory to local register
                bool active = leaf_mask[col];
                // write to shared memory
                sm_mask[offset] = active;
                if (active) {
#pragma unroll
                    for (uint32_t j = 0; j < DIM; j++) {
                        sm_data[offset * DIM + j] = leaf_data[col * DIM + j];
                    }

                    // write out the data if this node is selected (big leaf node)
                    if (sm_data[offset * DIM] > cut) {
                        selected_mask[col] = true;
#pragma unroll
                        for (uint32_t j = 0; j < DIM; j++) {
                            selected_data[col * DIM + j] = sm_data[offset * DIM + j];
                        }
                        printf("[Final] thread %d: col %d, data[0] %f\n", threadIdx.x,
                               col, sm_data[offset * DIM]);
                    }
                }
                printf("[LOAD DATA] thread %d: col %d, offset %d, data[0] %f\n",
                       threadIdx.x, col, offset, sm_data[offset * DIM]);

            } else {
                // If the thread is out of range, set the mask to false and data to
                // zero.
                sm_mask[offset] = false;
#pragma unroll
                for (uint32_t j = 0; j < DIM; j++) {
                    sm_data[offset * DIM + j] = 0.f;
                }
            }
        }

        // Add the total value of all previous blocks to the first value of this
        // block.
        if (threadIdx.x == 0) {
#pragma unroll
            for (uint32_t i = 0; i < DIM; i++) {
                // A simple add op.
                sm_data[i] += block_total_data[i];
            }
            printf("[ADD TOTAL] thread %d: data[0] %f\n", threadIdx.x, sm_data[0]);
        }

        // Sync threads to make sure the total value is updated.
        __syncthreads();

        // Parallel reduction (up-sweep).
        for (uint32_t s = N_THREADS, d = 1; s >= 1; s /= BR, d *= BR) {
            uint32_t offset = (BR * threadIdx.x + 1) * d - 1;
            // accumulate to node [offset + (BR - 1) * d].
            bool valid = offset + (BR - 1) * d < n_leaf;
            if (threadIdx.x < s && valid) {

                float accum_data[DIM] = {0.f};
#pragma unroll
                for (uint32_t i = 0; i < DIM; i++) {
#pragma unroll
                    for (uint32_t j = 0; j < BR; j++) {
                        // A simple add op.
                        // add node [offset + j * d] to accum_data
                        accum_data[i] += sm_data[(offset + j * d) * DIM + i];

                        if (threadIdx.x < n_leaf) {
                            printf("[REDUCE] thread %d: add sm node %d to accum data "
                                   "living in node %d, "
                                   "accum_data[0] %f\n",
                                   threadIdx.x, offset + j * d, offset + (BR - 1) * d,
                                   accum_data[0]);
                        }
                    }
                }

                // cut condition.
                if (accum_data[0] <= cut) {
                    // If the accumulated data (parent) is smaller than the
                    // cut, all the children are smaller and useless. So we
                    // can set the output mask to false. Note we only skip the last
                    // child, as the last child is the one that will be
                    // overwritten by the parent.
                    for (uint32_t i = 0; i < BR - 1; i++) {
                        uint32_t col = block_col + offset + i * d;
                        if (col < n_leaf) {
                            selected_mask[col] = false;
                        }
                        printf("[Final] thread %d: col %d, set to false\n", threadIdx.x,
                               col);
                    }
                } else {
                    // If the accumulated data (parent) is larger than the cut, we want
                    // to write out the children which are smaller than the cut.
                    for (uint32_t i = 0; i < BR; i++) {
                        uint32_t offset_node = offset + i * d;
                        uint32_t col = block_col + offset_node;
                        if (col < n_leaf && sm_mask[offset_node] &&
                            sm_data[offset_node * DIM] < cut) {
                            selected_mask[col] = true;
#pragma unroll
                            for (uint32_t j = 0; j < DIM; j++) {
                                selected_data[col * DIM + j] =
                                    sm_data[offset_node * DIM + j];
                            }
                            printf("[Final] thread %d: col %d, data[0] %f\n",
                                   threadIdx.x, col, sm_data[offset_node * DIM]);
                        }
                    }
                }

                // write the accumulated data to the last child.
                for (uint32_t i = 0; i < DIM; i++) {
                    sm_data[(offset + (BR - 1) * d) * DIM + i] = accum_data[i];
                    // now the last node in the children is actually the parent.
                    // So we set its mask to true.
                    sm_mask[offset + (BR - 1) * d] = true;
                }
            }
            __syncthreads();
        }
    }
}

// Check if a number x is a power of another number b (i.e., x = b^n for some
// integer n)
bool is_power_of(uint32_t x, uint32_t b) {
    if (b <= 1 || x <= 0) {
        return false;
    }
    unsigned int temp = x; // Create a copy of x
    while (temp % b == 0) {
        temp /= b;
    }
    return temp == 1;
}

template <uint32_t DIM, uint32_t BR>
std::vector<torch::Tensor> tree_cut_internel(const torch::Tensor leaf_data,
                                             const torch::Tensor leaf_mask,
                                             const float cut) {
    // Check the input tensor
    TORCH_CHECK(leaf_data.dim() == 2, "leaf_data must be 2D tensor");
    TORCH_CHECK(leaf_mask.dim() == 1, "leaf_mask must be 1D tensor");
    TORCH_CHECK(
        leaf_data.size(0) == leaf_mask.size(0),
        "leaf_data and leaf_mask must have the same size on the first dimension");
    TORCH_CHECK(leaf_data.size(1) == DIM, "leaf_data shape mismatch: expected ", DIM,
                ", got ", leaf_data.size(1));

    // Get the number of leaf nodes
    const uint32_t n_leaf = leaf_data.size(0);
    TORCH_CHECK(is_power_of(n_leaf, BR), "n_leaf must be a power of ", BR, " but got ",
                n_leaf);
    TORCH_CHECK(is_power_of(N_THREADS, BR), "N_THREADS must be a power of ", BR,
                " but got ", N_THREADS);

    // Allocate the output tensor
    torch::Tensor selected_data = torch::zeros({n_leaf, DIM}, leaf_data.options());
    torch::Tensor selected_mask = torch::zeros({n_leaf}, leaf_mask.options());

    // Call the kernel
    if (n_leaf) {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        const uint32_t shared_mem =
            N_THREADS * BR * (DIM * sizeof(float) + sizeof(bool));
        // Each thread process BR leaf nodes. So the number of threads in total
        // is n_leaf / BR.
        // Each block contains N_THREADS threads. So the shared memory that each
        // block requests is N_THREADS * BR * <size of each node>.
        tree_cut2_kernel<DIM, BR><<<(n_leaf / BR + N_THREADS - 1) / N_THREADS,
                                    N_THREADS, shared_mem, stream>>>(
            n_leaf, leaf_data.data_ptr<float>(), leaf_mask.data_ptr<bool>(), cut,
            selected_data.data_ptr<float>(), selected_mask.data_ptr<bool>());
    }

    // Return the output tensors
    auto outputs = std::vector<torch::Tensor>(2);
    outputs[0] = selected_data;
    outputs[1] = selected_mask;
    return outputs;
}

std::vector<torch::Tensor> tree_cut_tensor(const torch::Tensor leaf_data,
                                           const torch::Tensor leaf_mask,
                                           const uint32_t branch_factor,
                                           const float cut) {
    switch (branch_factor) {
    case 2:
        return tree_cut_internel<1, 2>(leaf_data, leaf_mask, cut);
    case 8:
        return tree_cut_internel<1, 8>(leaf_data, leaf_mask, cut);
    default:
        TORCH_CHECK(false, "Unsupported branch factor");
    }
}