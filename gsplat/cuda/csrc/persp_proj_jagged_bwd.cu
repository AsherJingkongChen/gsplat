#include "hip/hip_runtime.h"
#include "bindings.h"
#include "helpers.cuh"
#include "utils.cuh"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace gsplat {

namespace cg = cooperative_groups;

/****************************************************************************
 * Perspective Projection Backward Pass (Jagged Version)
 ****************************************************************************/

template <typename T>
__global__ void persp_proj_jagged_bwd_kernel(
    const uint32_t B,
    const int64_t nnz,
    const int64_t *__restrict__ g_sizes,  // [B]
    const int64_t *__restrict__ c_sizes,  // [B]
    const int64_t *__restrict__ g_indptr, // [B] start indices
    const int64_t *__restrict__ c_indptr, // [B] start indices
    const int64_t *__restrict__ n_indptr, // [B] start indices
    const T *__restrict__ means,          // [ggz, 3]
    const T *__restrict__ covars,         // [ggz, 3, 3]
    const T *__restrict__ Ks,             // [ccz, 3, 3]
    const uint32_t width,
    const uint32_t height,
    const T *__restrict__ v_means2d,  // [nnz, 2]
    const T *__restrict__ v_covars2d, // [nnz, 2, 2]
    T *__restrict__ v_means,          // [ggz, 3]
    T *__restrict__ v_covars          // [ggz, 3, 3]
) {

    // For now we'll upcast float16 and bfloat16 to float32
    using OpT = typename OpType<T>::type;

    // parallelize over nnz.
    uint32_t idx = cg::this_grid().thread_rank();
    if (idx >= nnz) {
        return;
    }

    // TODO: too many global memory accesses.
    int64_t bid =
        bin_search(n_indptr, B, static_cast<int64_t>(idx)); // batch id
    int64_t idx_local = idx - n_indptr[bid]; // local elem idx within Ci * Ni
    int64_t cid_local = idx_local / g_sizes[bid]; // local camera id within Ci
    int64_t gid_local = idx_local % g_sizes[bid]; // local gaussian id within Ni
    int64_t cid = cid_local + c_indptr[bid];      // camera id
    int64_t gid = gid_local + g_indptr[bid];      // gaussian id

    // shift pointers to the current camera and gaussian
    means += idx * 3;
    covars += idx * 9;
    v_means += idx * 3;
    v_covars += idx * 9;
    Ks += cid * 9;
    v_means2d += idx * 2;
    v_covars2d += idx * 4;

    OpT fx = Ks[0], cx = Ks[2], fy = Ks[4], cy = Ks[5];
    mat3<OpT> v_covar(0.f);
    vec3<OpT> v_mean(0.f);
    const vec3<OpT> mean = glm::make_vec3(means);
    const mat3<OpT> covar = glm::make_mat3(covars);
    const vec2<OpT> v_mean2d = glm::make_vec2(v_means2d);
    const mat2<OpT> v_covar2d = glm::make_mat2(v_covars2d);
    persp_proj_vjp<OpT>(
        mean,
        covar,
        fx,
        fy,
        cx,
        cy,
        width,
        height,
        glm::transpose(v_covar2d),
        v_mean2d,
        v_mean,
        v_covar
    );

    // write to outputs: glm is column-major but we want row-major
    GSPLAT_PRAGMA_UNROLL
    for (uint32_t i = 0; i < 3; i++) { // rows
        GSPLAT_PRAGMA_UNROLL
        for (uint32_t j = 0; j < 3; j++) { // cols
            v_covars[i * 3 + j] = T(v_covar[j][i]);
        }
    }

    GSPLAT_PRAGMA_UNROLL
    for (uint32_t i = 0; i < 3; i++) {
        v_means[i] = T(v_mean[i]);
    }
}

std::tuple<torch::Tensor, torch::Tensor> persp_proj_jagged_bwd_tensor(
    const torch::Tensor &g_sizes, // [B] gaussian sizes
    const torch::Tensor &means,   // [ggz, 3]
    const torch::Tensor &covars,  // [ggz, 3, 3]
    const torch::Tensor &c_sizes, // [B] camera sizes
    const torch::Tensor &Ks,      // [ccz, 3, 3]
    const uint32_t width,
    const uint32_t height,
    const torch::Tensor &v_means2d, // [nnz, 2]
    const torch::Tensor &v_covars2d // [nnz, 2, 2]
) {
    GSPLAT_DEVICE_GUARD(means);
    GSPLAT_CHECK_INPUT(g_sizes);
    GSPLAT_CHECK_INPUT(means);
    GSPLAT_CHECK_INPUT(covars);
    GSPLAT_CHECK_INPUT(c_sizes);
    GSPLAT_CHECK_INPUT(Ks);
    GSPLAT_CHECK_INPUT(v_means2d);
    GSPLAT_CHECK_INPUT(v_covars2d);

    // TODO: use inclusive sum
    uint32_t B = g_sizes.size(0);
    int64_t nnz = v_means2d.size(0);
    torch::Tensor c_indptr = torch::cumsum(c_sizes, 0, torch::kInt64) - c_sizes;
    torch::Tensor g_indptr = torch::cumsum(g_sizes, 0, torch::kInt64) - g_sizes;
    torch::Tensor n_sizes = c_sizes * g_sizes; // element size = Ci * Ni
    torch::Tensor n_indptr = torch::cumsum(n_sizes, 0, torch::kInt64) - n_sizes;

    torch::Tensor v_means = torch::zeros_like(means);
    torch::Tensor v_covars = torch::zeros_like(covars);

    if (nnz) {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        AT_DISPATCH_FLOATING_TYPES_AND2(
            at::ScalarType::Half,
            at::ScalarType::BFloat16,
            v_means.scalar_type(),
            "persp_proj_jagged_bwd",
            [&]() {
                persp_proj_jagged_bwd_kernel<scalar_t>
                    <<<(nnz + GSPLAT_N_THREADS - 1) / GSPLAT_N_THREADS,
                       GSPLAT_N_THREADS,
                       0,
                       stream>>>(
                        B,
                        nnz,
                        g_sizes.data_ptr<int64_t>(),
                        c_sizes.data_ptr<int64_t>(),
                        g_indptr.data_ptr<int64_t>(),
                        c_indptr.data_ptr<int64_t>(),
                        n_indptr.data_ptr<int64_t>(),
                        means.data_ptr<scalar_t>(),
                        covars.data_ptr<scalar_t>(),
                        Ks.data_ptr<scalar_t>(),
                        width,
                        height,
                        v_means2d.data_ptr<scalar_t>(),
                        v_covars2d.data_ptr<scalar_t>(),
                        v_means.data_ptr<scalar_t>(),
                        v_covars.data_ptr<scalar_t>()
                    );
            }
        );
    }
    return std::make_tuple(v_means, v_covars);
}

} // namespace gsplat