#include "hip/hip_runtime.h"
#include "bindings.h"
#include "helpers.cuh"
#include <hip/hip_cooperative_groups.h>
#include <hipcub/hipcub.hpp>

namespace cg = cooperative_groups;

/****************************************************************************
 * Gaussian Tile Intersection
 ****************************************************************************/

__global__ void isect_tiles(
    // if the data is [C, N, ...] or [nnz, ...] (packed)
    const bool packed,
    // parallelize over C * N, only used if packed is False
    const int C, const int N,
    // parallelize over nnz, only used if packed is True
    const int nnz,
    const int32_t *__restrict__ rindices, // [nnz] optional
    const int32_t *__restrict__ cindices, // [nnz] optional
    // data
    const float2 *__restrict__ means2d,              // [C, N, 2] or [nnz, 2]
    const int32_t *__restrict__ radii,               // [C, N] or [nnz]
    const float *__restrict__ depths,                // [C, N] or [nnz]
    const int64_t *__restrict__ cum_tiles_per_gauss, // [C, N] or [nnz]
    const int tile_size, const int tile_width, const int tile_height,
    const int tile_n_bits,
    int32_t *__restrict__ tiles_per_gauss, // [C, N] or [nnz]
    int64_t *__restrict__ isect_ids,       // [n_isects]
    int32_t *__restrict__ gauss_ids        // [n_isects]
) {
    // parallelize over C * N.
    unsigned idx = cg::this_grid().thread_rank();
    bool first_pass = cum_tiles_per_gauss == nullptr;
    if (idx >= (packed ? nnz : C * N))
        return;
    if (radii[idx] <= 0) {
        if (first_pass)
            tiles_per_gauss[idx] = 0;
        return;
    }

    float tile_radius = radii[idx] / static_cast<float>(tile_size);
    float tile_x = means2d[idx].x / tile_size;
    float tile_y = means2d[idx].y / tile_size;

    // tile_min is inclusive, tile_max is exclusive
    uint2 tile_min, tile_max;
    tile_min.x = min(max(0, (int)floor(tile_x - tile_radius)), tile_width);
    tile_min.y = min(max(0, (int)floor(tile_y - tile_radius)), tile_height);
    tile_max.x = min(max(0, (int)ceil(tile_x + tile_radius)), tile_width);
    tile_max.y = min(max(0, (int)ceil(tile_y + tile_radius)), tile_height);

    if (first_pass) {
        // first pass only writes out tiles_per_gauss
        tiles_per_gauss[idx] = (tile_max.y - tile_min.y) * (tile_max.x - tile_min.x);
        return;
    }

    int64_t cid; // camera id
    int32_t gid; // gaussian id
    if (packed) {
        // parallelize over nnz
        cid = rindices[idx];
        gid = cindices[idx];
    } else {
        // parallelize over C * N
        cid = idx / N;
        gid = idx % N;
    }
    const int64_t cid_enc = cid << (32 + tile_n_bits);

    int64_t depth_id_enc = (int64_t) * (int32_t *)&(depths[idx]);
    int64_t cur_idx = (idx == 0) ? 0 : cum_tiles_per_gauss[idx - 1];
    for (int i = tile_min.y; i < tile_max.y; ++i) {
        for (int j = tile_min.x; j < tile_max.x; ++j) {
            int64_t tile_id = i * tile_width + j;
            // e.g. tile_n_bits = 22:
            // camera id (10 bits) | tile id (22 bits) | depth (32 bits)
            isect_ids[cur_idx] = cid_enc | (tile_id << 32) | depth_id_enc;
            gauss_ids[cur_idx] = packed ? idx : gid;
            ++cur_idx;
        }
    }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor>
isect_tiles_tensor(const torch::Tensor &means2d,                // [C, N, 2] or [nnz, 2]
                   const torch::Tensor &radii,                  // [C, N] or [nnz]
                   const torch::Tensor &depths,                 // [C, N] or [nnz]
                   const at::optional<torch::Tensor> &rindices, // [nnz]
                   const at::optional<torch::Tensor> &cindices, // [nnz]
                   const int C, const int tile_size, const int tile_width,
                   const int tile_height, const bool sort) {
    DEVICE_GUARD(means2d);
    CHECK_INPUT(means2d);
    CHECK_INPUT(radii);
    CHECK_INPUT(depths);
    if (rindices.has_value()) {
        CHECK_INPUT(rindices.value());
    }
    if (cindices.has_value()) {
        CHECK_INPUT(cindices.value());
    }
    bool packed = means2d.dim() == 2;

    int N, nnz, totel_elems;
    int32_t *rindices_ptr;
    int32_t *cindices_ptr;
    if (packed) {
        nnz = means2d.size(0);
        totel_elems = nnz;
        assert(rindices.has_value() && cindices.has_value());
        rindices_ptr = rindices.value().data_ptr<int32_t>();
        cindices_ptr = cindices.value().data_ptr<int32_t>();
    } else {
        N = means2d.size(1); // number of gaussians
        totel_elems = C * N;
    }

    int n_tiles = tile_width * tile_height;
    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    // the number of bits needed to encode the camera id and tile id
    // Note: std::bit_width requires C++20
    // int tile_n_bits = std::bit_width(n_tiles);
    // int cam_n_bits = std::bit_width(C);
    int tile_n_bits = (int)floor(log2(n_tiles)) + 1;
    int cam_n_bits = (int)floor(log2(C)) + 1;
    // the first 32 bits are used for the camera id and tile id altogether, so check if
    // we have enough bits for them.
    assert(tile_n_bits + cam_n_bits <= 32);

    // first pass: compute number of tiles per gaussian
    torch::Tensor tiles_per_gauss =
        torch::empty_like(depths, depths.options().dtype(torch::kInt32));

    int64_t n_isects;
    torch::Tensor cum_tiles_per_gauss;
    if (totel_elems) {
        isect_tiles<<<(totel_elems + N_THREADS - 1) / N_THREADS, N_THREADS, 0,
                      stream>>>(
            packed, C, N, nnz, rindices_ptr, cindices_ptr,
            (float2 *)means2d.data_ptr<float>(), radii.data_ptr<int32_t>(),
            depths.data_ptr<float>(), nullptr, tile_size, tile_width, tile_height,
            tile_n_bits, tiles_per_gauss.data_ptr<int32_t>(), nullptr, nullptr);
        cum_tiles_per_gauss = torch::cumsum(tiles_per_gauss.view({-1}), 0);
        n_isects = cum_tiles_per_gauss[-1].item<int64_t>();
    } else {
        n_isects = 0;
    }

    // second pass: compute isect_ids and gauss_ids as a packed tensor
    torch::Tensor isect_ids =
        torch::empty({n_isects}, depths.options().dtype(torch::kInt64));
    torch::Tensor gauss_ids =
        torch::empty({n_isects}, depths.options().dtype(torch::kInt32));
    if (n_isects) {
        isect_tiles<<<(totel_elems + N_THREADS - 1) / N_THREADS, N_THREADS, 0,
                      stream>>>(
            packed, C, N, nnz, rindices_ptr, cindices_ptr,
            (float2 *)means2d.data_ptr<float>(), radii.data_ptr<int32_t>(),
            depths.data_ptr<float>(), cum_tiles_per_gauss.data_ptr<int64_t>(),
            tile_size, tile_width, tile_height, tile_n_bits, nullptr,
            isect_ids.data_ptr<int64_t>(), gauss_ids.data_ptr<int32_t>());
    }

    // optionally sort the Gaussians by isect_ids
    if (n_isects && sort) {
        torch::Tensor isect_ids_sorted = torch::empty_like(isect_ids);
        torch::Tensor gauss_ids_sorted = torch::empty_like(gauss_ids);
        CUB_WRAPPER(hipcub::DeviceRadixSort::SortPairs, isect_ids.data_ptr<int64_t>(),
                    isect_ids_sorted.data_ptr<int64_t>(), gauss_ids.data_ptr<int32_t>(),
                    gauss_ids_sorted.data_ptr<int32_t>(), n_isects, 0,
                    32 + tile_n_bits + cam_n_bits, stream);
        return std::make_tuple(tiles_per_gauss, isect_ids_sorted, gauss_ids_sorted);
    } else {
        return std::make_tuple(tiles_per_gauss, isect_ids, gauss_ids);
    }
}

__global__ void isect_offset_encode(const int n_isects,
                                    const int64_t *__restrict__ isect_ids, const int C,
                                    const int n_tiles, const int tile_n_bits,
                                    int32_t *__restrict__ offsets // [C, n_tiles]
) {
    // e.g., ids: [1, 1, 1, 3, 3], n_tiles = 6
    // counts: [0, 3, 0, 2, 0, 0]
    // cumsum: [0, 3, 3, 5, 5, 5]
    // offsets: [0, 0, 3, 3, 5, 5]
    unsigned idx = cg::this_grid().thread_rank();
    if (idx >= n_isects)
        return;

    int64_t isect_id_curr = isect_ids[idx] >> 32;
    int64_t cid_curr = isect_id_curr >> tile_n_bits;
    int64_t tid_curr = isect_id_curr & ((1 << tile_n_bits) - 1);
    int64_t id_curr = cid_curr * n_tiles + tid_curr;

    if (idx == 0) {
        // write out the offsets until the first valid tile (inclusive)
        for (int i = 0; i < id_curr + 1; ++i)
            offsets[i] = idx;
    }
    if (idx == n_isects - 1) {
        // write out the rest of the offsets
        for (int i = id_curr + 1; i < C * n_tiles; ++i)
            offsets[i] = n_isects;
    }

    if (idx > 0) {
        // visit the current and previous isect_id and check if the (cid, tile_id)
        // pair changes.
        int64_t isect_id_prev = isect_ids[idx - 1] >> 32; // shift out the depth
        if (isect_id_prev == isect_id_curr)
            return;

        // write out the offsets between the previous and current tiles
        int64_t cid_prev = isect_id_prev >> tile_n_bits;
        int64_t tid_prev = isect_id_prev & ((1 << tile_n_bits) - 1);
        int64_t id_prev = cid_prev * n_tiles + tid_prev;
        for (int i = id_prev + 1; i < id_curr + 1; ++i)
            offsets[i] = idx;
    }
}

torch::Tensor isect_offset_encode_tensor(const torch::Tensor &isect_ids, // [n_isects]
                                         const int C, const int tile_width,
                                         const int tile_height) {
    DEVICE_GUARD(isect_ids);
    CHECK_INPUT(isect_ids);

    int n_isects = isect_ids.size(0);
    torch::Tensor offsets = torch::empty({C, tile_height, tile_width},
                                         isect_ids.options().dtype(torch::kInt32));
    if (n_isects) {
        int n_tiles = tile_width * tile_height;
        int tile_n_bits = (int)floor(log2(n_tiles)) + 1;
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        isect_offset_encode<<<(n_isects + N_THREADS - 1) / N_THREADS, N_THREADS, 0,
                              stream>>>(n_isects, isect_ids.data_ptr<int64_t>(), C,
                                        n_tiles, tile_n_bits,
                                        offsets.data_ptr<int32_t>());
    } else {
        offsets.fill_(0);
    }
    return offsets;
}

/****************************************************************************
 * Rasterization
 ****************************************************************************/

__global__ void rasterize_to_indices_iter_kernel(
    const int step0, const int step1, const int C, const int N, const int n_isects,
    const float2 *__restrict__ means2d,  // [C, N, 2]
    const float3 *__restrict__ conics,   // [C, N, 3]
    const float *__restrict__ opacities, // [C, N]
    const int image_width, const int image_height, const int tile_size,
    const int tile_width, const int tile_height,
    const int32_t *__restrict__ tile_offsets, // [C, tile_height, tile_width]
    const int32_t *__restrict__ gauss_ids,    // [n_isects]
    const float *__restrict__ transmittances, // [C, image_height, image_width]
    const int32_t *__restrict__ chunk_starts, // [C, image_height, image_width]
    int32_t *__restrict__ chunk_cnts,         // [C, image_height, image_width]
    int32_t *__restrict__ out_gauss_ids,      // [n_elems]
    int32_t *__restrict__ out_pixel_ids       // [n_elems]
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t camera_id = block.group_index().x;
    int32_t tile_id = block.group_index().y * tile_width + block.group_index().z;
    unsigned i = block.group_index().y * tile_size + block.thread_index().y;
    unsigned j = block.group_index().z * tile_size + block.thread_index().x;

    // move pointers to the current camera
    means2d += camera_id * N;
    conics += camera_id * N;
    opacities += camera_id * N;
    tile_offsets += camera_id * tile_height * tile_width;
    transmittances += camera_id * image_height * image_width;

    float px = (float)j + 0.5f;
    float py = (float)i + 0.5f;
    int32_t pix_id = i * image_width + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < image_height && j < image_width);
    bool done = !inside;

    bool first_pass = chunk_starts == nullptr;
    int base;
    if (!first_pass && inside) {
        chunk_starts += camera_id * image_height * image_width;
        base = chunk_starts[pix_id];
    }

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int32_t range_start = tile_offsets[tile_id];
    int32_t range_end =
        (camera_id == C - 1) && (tile_id == tile_width * tile_height - 1)
            ? n_isects
            : tile_offsets[tile_id + 1];
    const int block_size = block.size();
    int num_batches = (range_end - range_start + block_size - 1) / block_size;

    if (step0 >= num_batches) {
        // this entire tile has been processed in the previous iterations
        // so we don't need to do anything.
        return;
    }

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ float3 xy_opacity_batch[MAX_BLOCK_SIZE];
    __shared__ float3 conic_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    // transmittance is gonna be used in the backward pass which requires a high
    // numerical precision so we (should) use double for it. However double make bwd
    // 1.5x slower so we stick with float for now.
    float T, next_T;
    if (inside) {
        T = transmittances[pix_id];
        next_T = T;
    }
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();

    int cnt = 0;
    for (int b = step0; b < min(step1, num_batches); ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range_start + block_size * b;
        int idx = batch_start + tr;
        if (idx < range_end) {
            int32_t g_id = gauss_ids[idx];
            id_batch[tr] = g_id;
            const float2 xy = means2d[g_id];
            const float opac = opacities[g_id];
            xy_opacity_batch[tr] = {xy.x, xy.y, opac};
            conic_batch[tr] = conics[g_id];
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range_end - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            const float3 conic = conic_batch[t];
            const float3 xy_opac = xy_opacity_batch[t];
            const float opac = xy_opac.z;
            const float2 delta = {xy_opac.x - px, xy_opac.y - py};
            const float sigma =
                0.5f * (conic.x * delta.x * delta.x + conic.z * delta.y * delta.y) +
                conic.y * delta.x * delta.y;
            float alpha = min(0.999f, opac * __expf(-sigma));

            if (sigma < 0.f || alpha < 1.f / 255.f) {
                continue;
            }

            next_T = T * (1.0f - alpha);
            if (next_T <= 1e-4) { // this pixel is done: exclusive
                done = true;
                break;
            }

            if (first_pass) {
                // First pass of this function we count the number of gaussians
                // that contribute to each pixel
                cnt += 1;
            } else {
                // Second pass we write out the gaussian ids and pixel ids
                int32_t g = id_batch[t];
                out_gauss_ids[base + cnt] = g;
                out_pixel_ids[base + cnt] =
                    pix_id + camera_id * image_height * image_width;
                cnt += 1;
            }

            T = next_T;
        }
    }

    if (inside && first_pass) {
        chunk_cnts += camera_id * image_height * image_width;
        chunk_cnts[pix_id] = cnt;
    }
}

std::tuple<torch::Tensor, torch::Tensor> rasterize_to_indices_iter_tensor(
    const int step0, const int step1,   // iteration steps
    const torch::Tensor transmittances, // [C, image_height, image_width]
    // Gaussian parameters
    const torch::Tensor &means2d,   // [C, N, 2]
    const torch::Tensor &conics,    // [C, N, 3]
    const torch::Tensor &opacities, // [C, N]
    // image size
    const int image_width, const int image_height, const int tile_size,
    // intersections
    const torch::Tensor &tile_offsets, // [C, tile_height, tile_width]
    const torch::Tensor &gauss_ids     // [n_isects]
) {
    DEVICE_GUARD(means2d);
    CHECK_INPUT(means2d);
    CHECK_INPUT(conics);
    CHECK_INPUT(opacities);
    CHECK_INPUT(tile_offsets);
    CHECK_INPUT(gauss_ids);

    int C = means2d.size(0); // number of cameras
    int N = means2d.size(1); // number of gaussians
    int tile_height = tile_offsets.size(1);
    int tile_width = tile_offsets.size(2);
    int n_isects = gauss_ids.size(0);

    // Each block covers a tile on the image. In total there are
    // C * tile_height * tile_width blocks.
    dim3 threads = {tile_size, tile_size, 1};
    dim3 blocks = {C, tile_height, tile_width};

    // First pass: count the number of gaussians that contribute to each pixel
    int64_t n_elems;
    torch::Tensor chunk_starts;
    if (n_isects) {
        torch::Tensor chunk_cnts = torch::zeros({C * image_height * image_width},
                                                means2d.options().dtype(torch::kInt32));
        rasterize_to_indices_iter_kernel<<<blocks, threads>>>(
            step0, step1, C, N, n_isects, (float2 *)means2d.data_ptr<float>(),
            (float3 *)conics.data_ptr<float>(), opacities.data_ptr<float>(),
            image_width, image_height, tile_size, tile_width, tile_height,
            tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
            transmittances.data_ptr<float>(), nullptr, chunk_cnts.data_ptr<int32_t>(),
            nullptr, nullptr);

        torch::Tensor cumsum = torch::cumsum(chunk_cnts, 0, chunk_cnts.scalar_type());
        n_elems = cumsum[-1].item<int64_t>();
        chunk_starts = cumsum - chunk_cnts;
    } else {
        n_elems = 0;
    }

    // Second pass: allocate memory and write out the gaussian and pixel ids.
    torch::Tensor out_gauss_ids =
        torch::empty({n_elems}, means2d.options().dtype(torch::kInt32));
    torch::Tensor out_pixel_ids =
        torch::empty({n_elems}, means2d.options().dtype(torch::kInt32));
    if (n_elems) {
        rasterize_to_indices_iter_kernel<<<blocks, threads>>>(
            step0, step1, C, N, n_isects, (float2 *)means2d.data_ptr<float>(),
            (float3 *)conics.data_ptr<float>(), opacities.data_ptr<float>(),
            image_width, image_height, tile_size, tile_width, tile_height,
            tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
            transmittances.data_ptr<float>(), chunk_starts.data_ptr<int32_t>(), nullptr,
            out_gauss_ids.data_ptr<int32_t>(), out_pixel_ids.data_ptr<int32_t>());
    }
    return std::make_tuple(out_gauss_ids, out_pixel_ids);
}

template <uint32_t COLOR_DIM>
__global__ void rasterize_to_pixels_fwd_kernel(
    const int C, const int N, const int n_isects, const bool packed,
    const float2 *__restrict__ means2d,    // [C, N, 2] or [nnz, 2]
    const float3 *__restrict__ conics,     // [C, N, 3] or [nnz, 3]
    const float *__restrict__ colors,      // [C, N, COLOR_DIM] or [nnz, COLOR_DIM]
    const float *__restrict__ opacities,   // [C, N] or [nnz]
    const float *__restrict__ backgrounds, // [C, COLOR_DIM]
    const int image_width, const int image_height, const int tile_size,
    const int tile_width, const int tile_height,
    const int32_t *__restrict__ tile_offsets, // [C, tile_height, tile_width]
    const int32_t *__restrict__ gauss_ids,    // [n_isects]
    float *__restrict__ render_colors, // [C, image_height, image_width, COLOR_DIM]
    float *__restrict__ render_alphas, // [C, image_height, image_width, 1]
    int32_t *__restrict__ last_ids     // [C, image_height, image_width]
) {
    // each thread draws one pixel, but also timeshares caching gaussians in a
    // shared tile

    auto block = cg::this_thread_block();
    int32_t camera_id = block.group_index().x;
    int32_t tile_id = block.group_index().y * tile_width + block.group_index().z;
    unsigned i = block.group_index().y * tile_size + block.thread_index().y;
    unsigned j = block.group_index().z * tile_size + block.thread_index().x;

    if (!packed) {
        // the data is with shape [C, N, ...]
        // move pointers to the current camera
        means2d += camera_id * N;
        conics += camera_id * N;
        colors += camera_id * N * COLOR_DIM;
        opacities += camera_id * N;
    }
    tile_offsets += camera_id * tile_height * tile_width;
    render_colors += camera_id * image_height * image_width * COLOR_DIM;
    render_alphas += camera_id * image_height * image_width;
    last_ids += camera_id * image_height * image_width;
    if (backgrounds != nullptr) {
        backgrounds += camera_id * COLOR_DIM;
    }

    float px = (float)j + 0.5f;
    float py = (float)i + 0.5f;
    int32_t pix_id = i * image_width + j;

    // return if out of bounds
    // keep not rasterizing threads around for reading data
    bool inside = (i < image_height && j < image_width);
    bool done = !inside;

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int32_t range_start = tile_offsets[tile_id];
    int32_t range_end =
        (camera_id == C - 1) && (tile_id == tile_width * tile_height - 1)
            ? n_isects
            : tile_offsets[tile_id + 1];
    const int block_size = block.size();
    int num_batches = (range_end - range_start + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ float3 xy_opacity_batch[MAX_BLOCK_SIZE];
    __shared__ float3 conic_batch[MAX_BLOCK_SIZE];

    // current visibility left to render
    // transmittance is gonna be used in the backward pass which requires a high
    // numerical precision so we use double for it. However double make bwd 1.5x slower
    // so we stick with float for now.
    float T = 1.0f;
    // index of most recent gaussian to write to this thread's pixel
    int cur_idx = 0;

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing its
    // designated pixel
    int tr = block.thread_rank();

    float pix_out[COLOR_DIM] = {0.f};
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before beginning next batch
        // end early if entire tile is done
        if (__syncthreads_count(done) >= block_size) {
            break;
        }

        // each thread fetch 1 gaussian from front to back
        // index of gaussian to load
        int batch_start = range_start + block_size * b;
        int idx = batch_start + tr;
        if (idx < range_end) {
            // if packed, g is the index in the packed tensor [nnz],
            // otherwise it is the gaussian index in N gaussians.
            int32_t g = gauss_ids[idx];
            id_batch[tr] = g;
            const float2 xy = means2d[g];
            const float opac = opacities[g];
            xy_opacity_batch[tr] = {xy.x, xy.y, opac};
            conic_batch[tr] = conics[g];
        }

        // wait for other threads to collect the gaussians in batch
        block.sync();

        // process gaussians in the current batch for this pixel
        int batch_size = min(block_size, range_end - batch_start);
        for (int t = 0; (t < batch_size) && !done; ++t) {
            const float3 conic = conic_batch[t];
            const float3 xy_opac = xy_opacity_batch[t];
            const float opac = xy_opac.z;
            const float2 delta = {xy_opac.x - px, xy_opac.y - py};
            const float sigma =
                0.5f * (conic.x * delta.x * delta.x + conic.z * delta.y * delta.y) +
                conic.y * delta.x * delta.y;
            float alpha = min(0.999f, opac * __expf(-sigma));
            if (sigma < 0.f || alpha < 1.f / 255.f) {
                continue;
            }

            const float next_T = T * (1.0f - alpha);
            if (next_T <= 1e-4) { // this pixel is done: exclusive
                done = true;
                break;
            }

            int32_t g = id_batch[t];
            const float vis = alpha * T;
            const float *c_ptr = colors + g * COLOR_DIM;
            PRAGMA_UNROLL
            for (int k = 0; k < COLOR_DIM; ++k) {
                pix_out[k] += c_ptr[k] * vis;
            }
            cur_idx = batch_start + t;

            T = next_T;
        }
    }

    if (inside) {
        // Here T is the transmittance AFTER the last gaussian in this pixel.
        // We (should) store double precision as T would be used in backward pass and
        // it can be very small and causing large diff in gradients with float32.
        // However, double precision makes the backward pass 1.5x slower so we stick
        // with float for now.
        render_alphas[pix_id] = 1.0f - T;
        PRAGMA_UNROLL
        for (int k = 0; k < COLOR_DIM; ++k) {
            render_colors[pix_id * COLOR_DIM + k] =
                backgrounds == nullptr ? pix_out[k] : (pix_out[k] + T * backgrounds[k]);
        }
        // index in bin of last gaussian in this pixel
        last_ids[pix_id] = cur_idx;
    }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> rasterize_to_pixels_fwd_tensor(
    // Gaussian parameters
    const torch::Tensor &means2d,   // [C, N, 2] or [nnz, 2]
    const torch::Tensor &conics,    // [C, N, 3] or [nnz, 3]
    const torch::Tensor &colors,    // [C, N, channels] or [nnz, channels]
    const torch::Tensor &opacities, // [C, N]  or [nnz]
    const at::optional<torch::Tensor> &backgrounds, // [C, channels]
    // image size
    const int image_width, const int image_height, const int tile_size,
    // intersections
    const torch::Tensor &tile_offsets, // [C, tile_height, tile_width]
    const torch::Tensor &gauss_ids     // [n_isects]
) {
    DEVICE_GUARD(means2d);
    CHECK_INPUT(means2d);
    CHECK_INPUT(conics);
    CHECK_INPUT(colors);
    CHECK_INPUT(opacities);
    CHECK_INPUT(tile_offsets);
    CHECK_INPUT(gauss_ids);
    if (backgrounds.has_value()) {
        CHECK_INPUT(backgrounds.value());
    }
    bool packed = means2d.dim() == 2;

    int C = tile_offsets.size(0);          // number of cameras
    int N = packed ? -1 : means2d.size(1); // number of gaussians
    int channels = colors.size(-1);
    int tile_height = tile_offsets.size(1);
    int tile_width = tile_offsets.size(2);
    int n_isects = gauss_ids.size(0);

    // Each block covers a tile on the image. In total there are
    // C * tile_height * tile_width blocks.
    dim3 threads = {tile_size, tile_size, 1};
    dim3 blocks = {C, tile_height, tile_width};

    torch::Tensor renders = torch::empty({C, image_height, image_width, channels},
                                         means2d.options().dtype(torch::kFloat32));
    torch::Tensor alphas = torch::empty({C, image_height, image_width, 1},
                                        means2d.options().dtype(torch::kFloat32));
    torch::Tensor last_ids = torch::empty({C, image_height, image_width},
                                          means2d.options().dtype(torch::kInt32));

    at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();

    // TODO: an optimization can be done by passing the actual number of channels into
    // the kernel functions and avoid necessary global memory writes. This requires
    // moving the channel padding from python to C side.
    switch (channels) {
    case 1:
        rasterize_to_pixels_fwd_kernel<1><<<blocks, threads, 0, stream>>>(
            C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
            (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
            opacities.data_ptr<float>(),
            backgrounds.has_value() ? backgrounds.value().data_ptr<float>() : nullptr,
            image_width, image_height, tile_size, tile_width, tile_height,
            tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
            renders.data_ptr<float>(), alphas.data_ptr<float>(),
            last_ids.data_ptr<int32_t>());
        break;
    case 2:
        rasterize_to_pixels_fwd_kernel<2><<<blocks, threads, 0, stream>>>(
            C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
            (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
            opacities.data_ptr<float>(),
            backgrounds.has_value() ? backgrounds.value().data_ptr<float>() : nullptr,
            image_width, image_height, tile_size, tile_width, tile_height,
            tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
            renders.data_ptr<float>(), alphas.data_ptr<float>(),
            last_ids.data_ptr<int32_t>());
        break;
    case 3:
        rasterize_to_pixels_fwd_kernel<3><<<blocks, threads, 0, stream>>>(
            C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
            (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
            opacities.data_ptr<float>(),
            backgrounds.has_value() ? backgrounds.value().data_ptr<float>() : nullptr,
            image_width, image_height, tile_size, tile_width, tile_height,
            tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
            renders.data_ptr<float>(), alphas.data_ptr<float>(),
            last_ids.data_ptr<int32_t>());
        break;
    case 4:
        rasterize_to_pixels_fwd_kernel<4><<<blocks, threads, 0, stream>>>(
            C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
            (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
            opacities.data_ptr<float>(),
            backgrounds.has_value() ? backgrounds.value().data_ptr<float>() : nullptr,
            image_width, image_height, tile_size, tile_width, tile_height,
            tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
            renders.data_ptr<float>(), alphas.data_ptr<float>(),
            last_ids.data_ptr<int32_t>());
        break;
    case 8:
        rasterize_to_pixels_fwd_kernel<8><<<blocks, threads, 0, stream>>>(
            C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
            (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
            opacities.data_ptr<float>(),
            backgrounds.has_value() ? backgrounds.value().data_ptr<float>() : nullptr,
            image_width, image_height, tile_size, tile_width, tile_height,
            tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
            renders.data_ptr<float>(), alphas.data_ptr<float>(),
            last_ids.data_ptr<int32_t>());
        break;
    case 16:
        rasterize_to_pixels_fwd_kernel<16><<<blocks, threads, 0, stream>>>(
            C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
            (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
            opacities.data_ptr<float>(),
            backgrounds.has_value() ? backgrounds.value().data_ptr<float>() : nullptr,
            image_width, image_height, tile_size, tile_width, tile_height,
            tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
            renders.data_ptr<float>(), alphas.data_ptr<float>(),
            last_ids.data_ptr<int32_t>());
        break;
    case 32:
        rasterize_to_pixels_fwd_kernel<32><<<blocks, threads, 0, stream>>>(
            C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
            (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
            opacities.data_ptr<float>(),
            backgrounds.has_value() ? backgrounds.value().data_ptr<float>() : nullptr,
            image_width, image_height, tile_size, tile_width, tile_height,
            tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
            renders.data_ptr<float>(), alphas.data_ptr<float>(),
            last_ids.data_ptr<int32_t>());
        break;
    case 64:
        rasterize_to_pixels_fwd_kernel<64><<<blocks, threads, 0, stream>>>(
            C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
            (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
            opacities.data_ptr<float>(),
            backgrounds.has_value() ? backgrounds.value().data_ptr<float>() : nullptr,
            image_width, image_height, tile_size, tile_width, tile_height,
            tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
            renders.data_ptr<float>(), alphas.data_ptr<float>(),
            last_ids.data_ptr<int32_t>());
        break;
    case 128:
        rasterize_to_pixels_fwd_kernel<128><<<blocks, threads, 0, stream>>>(
            C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
            (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
            opacities.data_ptr<float>(),
            backgrounds.has_value() ? backgrounds.value().data_ptr<float>() : nullptr,
            image_width, image_height, tile_size, tile_width, tile_height,
            tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
            renders.data_ptr<float>(), alphas.data_ptr<float>(),
            last_ids.data_ptr<int32_t>());
        break;
    case 256:
        rasterize_to_pixels_fwd_kernel<256><<<blocks, threads, 0, stream>>>(
            C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
            (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
            opacities.data_ptr<float>(),
            backgrounds.has_value() ? backgrounds.value().data_ptr<float>() : nullptr,
            image_width, image_height, tile_size, tile_width, tile_height,
            tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
            renders.data_ptr<float>(), alphas.data_ptr<float>(),
            last_ids.data_ptr<int32_t>());
        break;
    case 512:
        rasterize_to_pixels_fwd_kernel<512><<<blocks, threads, 0, stream>>>(
            C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
            (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
            opacities.data_ptr<float>(),
            backgrounds.has_value() ? backgrounds.value().data_ptr<float>() : nullptr,
            image_width, image_height, tile_size, tile_width, tile_height,
            tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
            renders.data_ptr<float>(), alphas.data_ptr<float>(),
            last_ids.data_ptr<int32_t>());
        break;
    default:
        AT_ERROR("Unsupported number of channels: ", channels);
    }
    return std::make_tuple(renders, alphas, last_ids);
}

template <uint32_t COLOR_DIM>
__global__ void rasterize_to_pixels_bwd_kernel(
    const int C, const int N, const int n_isects, const bool packed,
    // fwd inputs
    const float2 *__restrict__ means2d,    // [C, N, 2] or [nnz, 2]
    const float3 *__restrict__ conics,     // [C, N, 3] or [nnz, 3]
    const float *__restrict__ colors,      // [C, N, COLOR_DIM] or [nnz, COLOR_DIM]
    const float *__restrict__ opacities,   // [C, N] or [nnz]
    const float *__restrict__ backgrounds, // [C, COLOR_DIM] or [nnz, COLOR_DIM]
    const int image_width, const int image_height, const int tile_size,
    const int tile_width, const int tile_height,
    const int32_t *__restrict__ tile_offsets, // [C, tile_height, tile_width]
    const int32_t *__restrict__ gauss_ids,    // [n_isects]
    // fwd outputs
    const float *__restrict__ render_alphas, // [C, image_height, image_width, 1]
    const int32_t *__restrict__ last_ids,    // [C, image_height, image_width]
    // grad outputs
    const float
        *__restrict__ v_render_colors, // [C, image_height, image_width, COLOR_DIM]
    const float *__restrict__ v_render_alphas, // [C, image_height, image_width, 1]
    // grad inputs
    float2 *__restrict__ v_means2d_abs, // [C, N, 2] or [nnz, 2]
    float2 *__restrict__ v_means2d,     // [C, N, 2] or [nnz, 2]
    float3 *__restrict__ v_conics,      // [C, N, 3] or [nnz, 3]
    float *__restrict__ v_colors,       // [C, N, COLOR_DIM] or [nnz, COLOR_DIM]
    float *__restrict__ v_opacities     // [C, N] or [nnz]
) {
    auto block = cg::this_thread_block();
    int32_t camera_id = block.group_index().x;
    int32_t tile_id = block.group_index().y * tile_width + block.group_index().z;
    unsigned i = block.group_index().y * tile_size + block.thread_index().y;
    unsigned j = block.group_index().z * tile_size + block.thread_index().x;

    if (!packed) {
        // the data is with shape [C, N, ...]
        // move pointers to the current camera
        means2d += camera_id * N;
        conics += camera_id * N;
        colors += camera_id * N * COLOR_DIM;
        opacities += camera_id * N;
        v_means2d += camera_id * N;
        v_conics += camera_id * N;
        v_colors += camera_id * N * COLOR_DIM;
        v_opacities += camera_id * N;
        if (v_means2d_abs != nullptr) {
            v_means2d_abs += camera_id * N;
        }
    }
    tile_offsets += camera_id * tile_height * tile_width;
    render_alphas += camera_id * image_height * image_width;
    last_ids += camera_id * image_height * image_width;
    v_render_colors += camera_id * image_height * image_width * COLOR_DIM;
    v_render_alphas += camera_id * image_height * image_width;
    if (backgrounds != nullptr) {
        backgrounds += camera_id * COLOR_DIM;
    }

    const float px = (float)j + 0.5f;
    const float py = (float)i + 0.5f;
    // clamp this value to the last pixel
    const int32_t pix_id = min(i * image_width + j, image_width * image_height - 1);

    // keep not rasterizing threads around for reading data
    bool inside = (i < image_height && j < image_width);

    // have all threads in tile process the same gaussians in batches
    // first collect gaussians between range.x and range.y in batches
    // which gaussians to look through in this tile
    int32_t range_start = tile_offsets[tile_id];
    int32_t range_end =
        (camera_id == C - 1) && (tile_id == tile_width * tile_height - 1)
            ? n_isects
            : tile_offsets[tile_id + 1];
    const int block_size = block.size();
    const int num_batches = (range_end - range_start + block_size - 1) / block_size;

    __shared__ int32_t id_batch[MAX_BLOCK_SIZE];
    __shared__ float3 xy_opacity_batch[MAX_BLOCK_SIZE];
    __shared__ float3 conic_batch[MAX_BLOCK_SIZE];
    __shared__ float rgbs_batch[MAX_BLOCK_SIZE * COLOR_DIM];

    // this is the T AFTER the last gaussian in this pixel
    float T_final = 1.0f - render_alphas[pix_id];
    float T = T_final;
    // the contribution from gaussians behind the current one
    float buffer[COLOR_DIM] = {0.f};
    // index of last gaussian to contribute to this pixel
    const int bin_final = inside ? last_ids[pix_id] : 0;

    // df/d_out for this pixel
    float v_render_c[COLOR_DIM];
    PRAGMA_UNROLL
    for (int k = 0; k < COLOR_DIM; ++k) {
        v_render_c[k] = v_render_colors[pix_id * COLOR_DIM + k];
    }
    const float v_render_a = v_render_alphas[pix_id];

    // collect and process batches of gaussians
    // each thread loads one gaussian at a time before rasterizing
    const int tr = block.thread_rank();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    const int warp_bin_final = cg::reduce(warp, bin_final, cg::greater<int>());
    for (int b = 0; b < num_batches; ++b) {
        // resync all threads before writing next batch of shared mem
        block.sync();

        // each thread fetch 1 gaussian from back to front
        // 0 index will be furthest back in batch
        // index of gaussian to load
        // batch end is the index of the last gaussian in the batch
        const int batch_end = range_end - 1 - block_size * b;
        int batch_size = min(block_size, batch_end + 1 - range_start);
        const int idx = batch_end - tr;
        if (idx >= range_start) {
            // if packed, g is the index in the packed tensor [nnz],
            // otherwise it is the gaussian index in N gaussians.
            int32_t g = gauss_ids[idx];
            id_batch[tr] = g;
            const float2 xy = means2d[g];
            const float opac = opacities[g];
            xy_opacity_batch[tr] = {xy.x, xy.y, opac};
            conic_batch[tr] = conics[g];
            PRAGMA_UNROLL
            for (int k = 0; k < COLOR_DIM; ++k) {
                rgbs_batch[tr * COLOR_DIM + k] = colors[g * COLOR_DIM + k];
            }
        }
        // wait for other threads to collect the gaussians in batch
        block.sync();
        // process gaussians in the current batch for this pixel
        // 0 index is the furthest back gaussian in the batch
        for (int t = max(0, batch_end - warp_bin_final); t < batch_size; ++t) {
            int valid = inside;
            if (batch_end - t > bin_final) {
                valid = 0;
            }
            float alpha;
            float opac;
            float2 delta;
            float3 conic;
            float vis;

            if (valid) {
                conic = conic_batch[t];
                float3 xy_opac = xy_opacity_batch[t];
                opac = xy_opac.z;
                delta = {xy_opac.x - px, xy_opac.y - py};
                float sigma =
                    0.5f * (conic.x * delta.x * delta.x + conic.z * delta.y * delta.y) +
                    conic.y * delta.x * delta.y;
                vis = __expf(-sigma);
                alpha = min(0.999f, opac * vis);
                if (sigma < 0.f || alpha < 1.f / 255.f) {
                    valid = 0;
                }
            }

            // if all threads are inactive in this warp, skip this loop
            if (!warp.any(valid)) {
                continue;
            }
            float v_rgb_local[COLOR_DIM] = {0.f};
            float3 v_conic_local = {0.f, 0.f, 0.f};
            float2 v_xy_local = {0.f, 0.f};
            float2 v_xy_abs_local = {0.f, 0.f};
            float v_opacity_local = 0.f;
            // initialize everything to 0, only set if the lane is valid
            if (valid) {
                // compute the current T for this gaussian
                float ra = 1.0f / (1.0f - alpha);
                T *= ra;
                // update v_rgb for this gaussian
                const float fac = alpha * T;
                PRAGMA_UNROLL
                for (int k = 0; k < COLOR_DIM; ++k) {
                    v_rgb_local[k] = fac * v_render_c[k];
                }
                // contribution from this pixel
                float v_alpha = 0.f;
                for (int k = 0; k < COLOR_DIM; ++k) {
                    v_alpha += (rgbs_batch[t * COLOR_DIM + k] * T - buffer[k] * ra) *
                               v_render_c[k];
                }

                v_alpha += T_final * ra * v_render_a;
                // contribution from background pixel
                if (backgrounds != nullptr) {
                    float accum = 0.f;
                    PRAGMA_UNROLL
                    for (int k = 0; k < COLOR_DIM; ++k) {
                        accum += backgrounds[k] * v_render_c[k];
                    }
                    v_alpha += -T_final * ra * accum;
                }

                if (opac * vis <= 0.999f) {
                    const float v_sigma = -opac * vis * v_alpha;
                    v_conic_local = {0.5f * v_sigma * delta.x * delta.x,
                                     v_sigma * delta.x * delta.y,
                                     0.5f * v_sigma * delta.y * delta.y};
                    v_xy_local = {v_sigma * (conic.x * delta.x + conic.y * delta.y),
                                  v_sigma * (conic.y * delta.x + conic.z * delta.y)};
                    if (v_means2d_abs != nullptr) {
                        v_xy_abs_local = {abs(v_xy_local.x), abs(v_xy_local.y)};
                    }
                    v_opacity_local = vis * v_alpha;
                }

                PRAGMA_UNROLL
                for (int k = 0; k < COLOR_DIM; ++k) {
                    buffer[k] += rgbs_batch[t * COLOR_DIM + k] * fac;
                }
            }
            warpSum<COLOR_DIM, float>(v_rgb_local, warp);
            warpSum(v_conic_local, warp);
            warpSum(v_xy_local, warp);
            if (v_means2d_abs != nullptr) {
                warpSum(v_xy_abs_local, warp);
            }
            warpSum(v_opacity_local, warp);
            if (warp.thread_rank() == 0) {
                int32_t g = id_batch[t];
                float *v_rgb_ptr = (float *)(v_colors) + COLOR_DIM * g;
                PRAGMA_UNROLL
                for (int k = 0; k < COLOR_DIM; ++k) {
                    atomicAdd(v_rgb_ptr + k, v_rgb_local[k]);
                }

                float *v_conic_ptr = (float *)(v_conics) + 3 * g;
                atomicAdd(v_conic_ptr, v_conic_local.x);
                atomicAdd(v_conic_ptr + 1, v_conic_local.y);
                atomicAdd(v_conic_ptr + 2, v_conic_local.z);

                float *v_xy_ptr = (float *)(v_means2d) + 2 * g;
                atomicAdd(v_xy_ptr, v_xy_local.x);
                atomicAdd(v_xy_ptr + 1, v_xy_local.y);

                if (v_means2d_abs != nullptr) {
                    float *v_xy_abs_ptr = (float *)(v_means2d_abs) + 2 * g;
                    atomicAdd(v_xy_abs_ptr, v_xy_abs_local.x);
                    atomicAdd(v_xy_abs_ptr + 1, v_xy_abs_local.y);
                }

                atomicAdd(v_opacities + g, v_opacity_local);
            }
        }
    }
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
rasterize_to_pixels_bwd_tensor(
    // Gaussian parameters
    const torch::Tensor &means2d,                   // [C, N, 2] or [nnz, 2]
    const torch::Tensor &conics,                    // [C, N, 3] or [nnz, 3]
    const torch::Tensor &colors,                    // [C, N, 3] or [nnz, 3]
    const torch::Tensor &opacities,                 // [C, N] or [nnz]
    const at::optional<torch::Tensor> &backgrounds, // [C, 3]
    // image size
    const int image_width, const int image_height, const int tile_size,
    // intersections
    const torch::Tensor &tile_offsets, // [C, tile_height, tile_width]
    const torch::Tensor &gauss_ids,    // [n_isects]
    // forward outputs
    const torch::Tensor &render_alphas, // [C, image_height, image_width, 1]
    const torch::Tensor &last_ids,      // [C, image_height, image_width]
    // gradients of outputs
    const torch::Tensor &v_render_colors, // [C, image_height, image_width, 3]
    const torch::Tensor &v_render_alphas, // [C, image_height, image_width, 1]
    // options
    bool compute_means2d_absgrad) {
    DEVICE_GUARD(means2d);
    CHECK_INPUT(means2d);
    CHECK_INPUT(conics);
    CHECK_INPUT(colors);
    CHECK_INPUT(opacities);
    CHECK_INPUT(tile_offsets);
    CHECK_INPUT(gauss_ids);
    CHECK_INPUT(render_alphas);
    CHECK_INPUT(last_ids);
    CHECK_INPUT(v_render_colors);
    CHECK_INPUT(v_render_alphas);
    if (backgrounds.has_value()) {
        CHECK_INPUT(backgrounds.value());
    }

    bool packed = means2d.dim() == 2;

    int C = tile_offsets.size(0);          // number of cameras
    int N = packed ? -1 : means2d.size(1); // number of gaussians
    int n_isects = gauss_ids.size(0);
    int COLOR_DIM = colors.size(-1);
    int tile_height = tile_offsets.size(1);
    int tile_width = tile_offsets.size(2);

    // Each block covers a tile on the image. In total there are
    // C * tile_height * tile_width blocks.
    dim3 threads = {tile_size, tile_size, 1};
    dim3 blocks = {C, tile_height, tile_width};

    torch::Tensor v_means2d = torch::zeros_like(means2d);
    torch::Tensor v_conics = torch::zeros_like(conics);
    torch::Tensor v_colors = torch::zeros_like(colors);
    torch::Tensor v_opacities = torch::zeros_like(opacities);
    torch::Tensor v_means2d_abs;
    if (compute_means2d_absgrad) {
        v_means2d_abs = torch::zeros_like(means2d);
    }

    if (n_isects) {
        at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
        switch (COLOR_DIM) {
        case 1:
            rasterize_to_pixels_bwd_kernel<1><<<blocks, threads, 0, stream>>>(
                C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
                (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
                opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                image_width, image_height, tile_size, tile_width, tile_height,
                tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
                render_alphas.data_ptr<float>(), last_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
                compute_means2d_absgrad ? (float2 *)v_means2d_abs.data_ptr<float>()
                                        : nullptr,
                (float2 *)v_means2d.data_ptr<float>(),
                (float3 *)v_conics.data_ptr<float>(), v_colors.data_ptr<float>(),
                v_opacities.data_ptr<float>());
            break;
        case 2:
            rasterize_to_pixels_bwd_kernel<2><<<blocks, threads, 0, stream>>>(
                C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
                (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
                opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                image_width, image_height, tile_size, tile_width, tile_height,
                tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
                render_alphas.data_ptr<float>(), last_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
                compute_means2d_absgrad ? (float2 *)v_means2d_abs.data_ptr<float>()
                                        : nullptr,
                (float2 *)v_means2d.data_ptr<float>(),
                (float3 *)v_conics.data_ptr<float>(), v_colors.data_ptr<float>(),
                v_opacities.data_ptr<float>());
            break;
        case 3:
            rasterize_to_pixels_bwd_kernel<3><<<blocks, threads, 0, stream>>>(
                C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
                (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
                opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                image_width, image_height, tile_size, tile_width, tile_height,
                tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
                render_alphas.data_ptr<float>(), last_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
                compute_means2d_absgrad ? (float2 *)v_means2d_abs.data_ptr<float>()
                                        : nullptr,
                (float2 *)v_means2d.data_ptr<float>(),
                (float3 *)v_conics.data_ptr<float>(), v_colors.data_ptr<float>(),
                v_opacities.data_ptr<float>());
            break;
        case 4:
            rasterize_to_pixels_bwd_kernel<4><<<blocks, threads, 0, stream>>>(
                C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
                (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
                opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                image_width, image_height, tile_size, tile_width, tile_height,
                tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
                render_alphas.data_ptr<float>(), last_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
                compute_means2d_absgrad ? (float2 *)v_means2d_abs.data_ptr<float>()
                                        : nullptr,
                (float2 *)v_means2d.data_ptr<float>(),
                (float3 *)v_conics.data_ptr<float>(), v_colors.data_ptr<float>(),
                v_opacities.data_ptr<float>());
            break;
        case 8:
            rasterize_to_pixels_bwd_kernel<8><<<blocks, threads, 0, stream>>>(
                C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
                (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
                opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                image_width, image_height, tile_size, tile_width, tile_height,
                tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
                render_alphas.data_ptr<float>(), last_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
                compute_means2d_absgrad ? (float2 *)v_means2d_abs.data_ptr<float>()
                                        : nullptr,
                (float2 *)v_means2d.data_ptr<float>(),
                (float3 *)v_conics.data_ptr<float>(), v_colors.data_ptr<float>(),
                v_opacities.data_ptr<float>());
            break;
        case 16:
            rasterize_to_pixels_bwd_kernel<16><<<blocks, threads, 0, stream>>>(
                C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
                (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
                opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                image_width, image_height, tile_size, tile_width, tile_height,
                tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
                render_alphas.data_ptr<float>(), last_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
                compute_means2d_absgrad ? (float2 *)v_means2d_abs.data_ptr<float>()
                                        : nullptr,
                (float2 *)v_means2d.data_ptr<float>(),
                (float3 *)v_conics.data_ptr<float>(), v_colors.data_ptr<float>(),
                v_opacities.data_ptr<float>());
            break;
        case 32:
            rasterize_to_pixels_bwd_kernel<32><<<blocks, threads, 0, stream>>>(
                C, N, n_isects, packed, (float2 *)means2d.data_ptr<float>(),
                (float3 *)conics.data_ptr<float>(), colors.data_ptr<float>(),
                opacities.data_ptr<float>(),
                backgrounds.has_value() ? backgrounds.value().data_ptr<float>()
                                        : nullptr,
                image_width, image_height, tile_size, tile_width, tile_height,
                tile_offsets.data_ptr<int32_t>(), gauss_ids.data_ptr<int32_t>(),
                render_alphas.data_ptr<float>(), last_ids.data_ptr<int32_t>(),
                v_render_colors.data_ptr<float>(), v_render_alphas.data_ptr<float>(),
                compute_means2d_absgrad ? (float2 *)v_means2d_abs.data_ptr<float>()
                                        : nullptr,
                (float2 *)v_means2d.data_ptr<float>(),
                (float3 *)v_conics.data_ptr<float>(), v_colors.data_ptr<float>(),
                v_opacities.data_ptr<float>());
            break;
        default:
            AT_ERROR("Unsupported number of channels: ", COLOR_DIM);
        }
    }

    return std::make_tuple(v_means2d_abs, v_means2d, v_conics, v_colors, v_opacities);
}
